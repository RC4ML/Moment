#include "cache.cuh"
#include "cache_impl.cuh"
#include <cstdio>      // 用于 fopen / fprintf / fclose
#include <cstdlib>     // 用于 malloc / free
#include <cstdint>     // 用于 int64_t
#include <cstring>     // 可选：strerror
#include <iostream>

bool dump_access_times_to_txt(const char* file_path,
                              const unsigned long long* data,
                              std::size_t num_nodes) {
    FILE* fp = std::fopen(file_path, "w");
    if (!fp) {
        std::cerr << "Failed to open file " << file_path
                  << " : " << std::strerror(errno) << '\n';
        return false;
    }

    for (std::size_t i = 0; i < num_nodes; ++i) {
        // 每行写一个十进制整数，末尾加换行
        if (std::fprintf(fp, "%llu\n", data[i]) < 0) {
            std::cerr << "Write error at index " << i << '\n';
            std::fclose(fp);
            return false;
        }
    }

    std::fclose(fp);
    return true;
}

bool destroy_file(const char* file_path) {
    if (std::remove(file_path) == 0) {          // 删除成功返回 0
        return true;
    } else {
        std::cerr << "Failed to remove " << file_path
                  << " : " << std::strerror(errno) << '\n';
        return false;
    }
}

class PreSCCacheController : public CacheController
{
public:
    PreSCCacheController(int32_t train_step, int32_t device_count)
    {
        train_step_ = train_step;
        device_count_ = device_count;
    }

    virtual ~PreSCCacheController() {}

    void Initialize(
        int32_t dev_id,
        int32_t total_num_nodes) override
    {
        device_idx_ = dev_id;
        total_num_nodes_ = total_num_nodes;
        hipSetDevice(dev_id);
        hipHostAlloc(&node_access_time_, int64_t(int64_t(total_num_nodes) * sizeof(unsigned long long int)), hipHostMallocMapped);
        hipMemset(node_access_time_, 0, int64_t(int64_t(total_num_nodes) * sizeof(unsigned long long int)));
        cudaCheckError();
        // hipMalloc(&edge_access_time_, int64_t(int64_t(total_num_nodes) * sizeof(unsigned long long int)));
        // hipMemset(edge_access_time_, 0, int64_t(int64_t(total_num_nodes) * sizeof(unsigned long long int)));
        // cudaCheckError();

        iter_ = 0;
        max_ids_ = 0;
        hipMalloc(&d_global_count_, 4);
        h_global_count_ = (int32_t *)malloc(4);
        find_iter_ = 0;
        h_cache_hit_ = 0;
    }

    void Finalize() override
    {
        // pos_map_->clear();
    }

    void CacheProfiling(
        int32_t *sampled_ids,
        int32_t *agg_src_id,
        int32_t *agg_dst_id,
        int32_t *agg_src_off,
        int32_t *agg_dst_off,
        int32_t *node_counter,
        int32_t *edge_counter,
        bool is_presc,
        void *stream) override
    {
        dim3 block_num(32, 1);
        dim3 thread_num(1024, 1);

        if (is_presc)
        {
            int32_t *h_node_counter = (int32_t *)malloc(16 * sizeof(int32_t));
            hipMemcpy(h_node_counter, node_counter, 64, hipMemcpyDeviceToHost);
            HotnessMeasure<<<block_num, thread_num, 0, static_cast<hipStream_t>(stream)>>>(sampled_ids, node_counter, node_access_time_);

            if (h_node_counter[INTRABATCH_CON * 2 + 1] > max_ids_)
            {
                max_ids_ = h_node_counter[INTRABATCH_CON * 2 + 1];
            }
            if (iter_ == (train_step_ - 1))
            {
                iter_ = 0;
            }
            free(h_node_counter);
        }
        iter_++;
    }

    /*num candidates = sampled num*/
    void InitializeMap(int node_capacity, int edge_capacity) override
    {
        hipSetDevice(device_idx_);
        node_capacity_ = node_capacity;
        edge_capacity_ = edge_capacity;

        auto invalid_key = CACHEMISS_FLAG;
        auto invalid_value = CACHEMISS_FLAG;

        node_map_ = new bght::bcht<int32_t, int32_t>(int64_t(node_capacity_ * device_count_) * 2, invalid_key, invalid_value);
        cudaCheckError();

        edge_index_map_ = new bght::bcht<int32_t, char>(int64_t(edge_capacity_ * device_count_) * 2, invalid_key, invalid_value);
        cudaCheckError();

        edge_offset_map_ = new bght::bcht<int32_t, int32_t>(int64_t(edge_capacity_ * device_count_) * 2, invalid_key, invalid_value);
        cudaCheckError();
    }

    void Insert(int32_t *QT, int32_t *QF, int32_t cache_expand, int32_t Kg) override
    {
        hipSetDevice(device_idx_);
        cudaCheckError();

        hipMalloc(&pair_, int64_t(int64_t(node_capacity_ * cache_expand) * sizeof(pair_type)));
        cudaCheckError();
        dim3 block_num(80, 1);
        dim3 thread_num(1024, 1);
        hipStream_t stream;
        hipStreamCreate(&stream);
        InitPair<<<block_num, thread_num>>>(pair_, QF, node_capacity_, cache_expand, Kg);
        cudaCheckError();
        node_map_->insert(pair_, (pair_ + node_capacity_ * cache_expand), stream);
        cudaCheckError();
        // if(success){
        //     std::cout<<"Feature Cache Successfully Initialized\n";
        // }
        hipDeviceSynchronize();
        cudaCheckError();
        hipFree(pair_);
        cudaCheckError();
        // hipFree(cache_ids_);
        // cudaCheckError();
        // hipFree(cache_offset_);
        // cudaCheckError();

        index_pair_type *index_pair;
        offset_pair_type *offset_pair;
        hipMalloc(&index_pair, int64_t(int64_t(edge_capacity_ * cache_expand) * sizeof(index_pair_type)));
        cudaCheckError();
        hipMalloc(&offset_pair, int64_t(int64_t(edge_capacity_ * cache_expand) * sizeof(offset_pair_type)));
        cudaCheckError();

        InitIndexPair<<<block_num, thread_num>>>(index_pair, QT, edge_capacity_, cache_expand, Kg, device_idx_ / Kg);
        InitOffsetPair<<<block_num, thread_num>>>(offset_pair, QT, edge_capacity_, cache_expand, Kg);

        edge_index_map_->insert(index_pair, (index_pair + edge_capacity_ * cache_expand), stream);
        cudaCheckError();

        edge_offset_map_->insert(offset_pair, (offset_pair + edge_capacity_ * cache_expand), stream);

        cudaCheckError();
        hipDeviceSynchronize();
        hipFree(index_pair);
        hipFree(offset_pair);
    }

    void HybridInsert(int32_t *QF, int32_t cpu_cache_capacity, int32_t gpu_cache_capacity) override
    { // only feature now
        hipSetDevice(device_idx_);
        cudaCheckError();

        hipMalloc(&pair_, int64_t(int64_t(cpu_cache_capacity + gpu_cache_capacity) * sizeof(pair_type)));
        cudaCheckError();
        dim3 block_num(80, 1);
        dim3 thread_num(1024, 1);
        hipStream_t stream;
        hipStreamCreate(&stream);
        HybridInitPair<<<block_num, thread_num>>>(pair_, QF, cpu_cache_capacity, gpu_cache_capacity);
        cudaCheckError();
        node_map_->insert(pair_, (pair_ + (cpu_cache_capacity + gpu_cache_capacity)), stream);
        cudaCheckError();
        // if(success){
        //     std::cout<<"Feature Cache Successfully Initialized\n";
        // }
        hipDeviceSynchronize();
        cudaCheckError();
        hipFree(pair_);
        cudaCheckError();
        // hipFree(cache_ids_);
        // cudaCheckError();
        // hipFree(cache_offset_);
        // cudaCheckError();
    }

    void AccessCount(
        int32_t *d_key,
        int32_t num_keys,
        void *stream) override
    {
    }

    unsigned long long int *GetNodeAccessedMap()
    {
        return node_access_time_;
        // return nullptr;
    }

    unsigned long long int *GetEdgeAccessedMap()
    {
        // return edge_access_time_;
        return nullptr;
    }

    void FindFeat(
        int32_t *sampled_ids,
        int32_t *cache_offset,
        int32_t *node_counter,
        int32_t op_id,
        void *stream) override
    {
        int32_t *h_node_counter = (int32_t *)malloc(64);
        hipMemcpy(h_node_counter, node_counter, 64, hipMemcpyDeviceToHost);

        int32_t node_off = h_node_counter[(op_id % INTRABATCH_CON) * 2];
        int32_t batch_size = h_node_counter[(op_id % INTRABATCH_CON) * 2 + 1];
        if (batch_size == 0)
        {
            std::cout << "invalid batchsize for feature extraction " << h_node_counter[(op_id % INTRABATCH_CON) * 2] << " " << h_node_counter[(op_id % INTRABATCH_CON) * 2 + 1] << "\n";
            return;
        }
        node_map_->find(sampled_ids + node_off, sampled_ids + (node_off + batch_size), cache_offset, static_cast<hipStream_t>(stream));
        // if(find_iter_ % 500 == 0){
        //     hipMemsetAsync(d_global_count_, 0, 4, static_cast<hipStream_t>(stream));
        //     dim3 block_num(48, 1);
        //     dim3 thread_num(1024, 1);
        //     feature_cache_hit<<<block_num, thread_num, 0, static_cast<hipStream_t>(stream)>>>(cache_offset, batch_size, d_global_count_);
        //     hipMemcpy(h_global_count_, d_global_count_, 4, hipMemcpyDeviceToHost);
        //     h_cache_hit_ += h_global_count_[0];
        //     if(op_id == 8){
        //         std::cout<<device_idx_<<" Feature Cache Hit: "<<(h_cache_hit_ * 1.0 / h_node_counter[INTRABATCH_CON * 2 + 1])<<std::endl;
        //         h_cache_hit_ = 0;
        //     }
        // }
        if (op_id == 8)
        {
            // std::cout<<device_idx_<<" Feature Cache Hit: "<<h_cache_hit_<<" "<<(h_cache_hit_ * 1.0 / h_node_counter[9])<<std::endl;
            // h_cache_hit_ = 0;
            find_iter_++;
            // std::cout<<"find_iter "<<find_iter_<<std::endl;
        }
    }

    void FindTopo(int32_t *input_ids,
                  char *partition_index,
                  int32_t *partition_offset,
                  int32_t batch_size,
                  int32_t op_id,
                  void *strm_hdl,
                  int32_t device_id) override
    {
        edge_index_map_->find(input_ids, input_ids + batch_size, partition_index, static_cast<hipStream_t>(strm_hdl));
        edge_offset_map_->find(input_ids, input_ids + batch_size, partition_offset, static_cast<hipStream_t>(strm_hdl));

        // if(find_iter_[device_id] % 500 == 0){
        //     hipMemsetAsync(d_global_count_[device_id], 0, 4, static_cast<hipStream_t>(strm_hdl));
        //     dim3 block_num(48, 1);
        //     dim3 thread_num(1024, 1);
        //     cache_hit<<<block_num, thread_num, 0, static_cast<hipStream_t>(strm_hdl)>>>(partition_index, batch_size, d_global_count_[device_id]);
        //     hipMemcpy(h_global_count_[device_id], d_global_count_[device_id], 4, hipMemcpyDeviceToHost);
        //     h_cache_hit_[device_id] += ((h_global_count_[device_id])[0]);
        //     h_batch_size_[device_id] += batch_size;
        //     if(op_id == 4){
        //         std::cout<<device_id<<" Topo Cache Hit: "<<h_cache_hit_[device_id]<<" "<<(h_cache_hit_[device_id] * 1.0 / h_batch_size_[device_id])<<std::endl;
        //         h_cache_hit_[device_id] = 0;
        //         h_batch_size_[device_id] = 0;
        //     }
        // }
        // if(op_id == 4){
        //     find_iter_[device_id] += 1;
        // }
    }

    int32_t MaxIdNum() override
    {
        return max_ids_;
    }

private:
    int32_t device_idx_;
    int32_t device_count_;
    int32_t total_num_nodes_;

    unsigned long long int *node_access_time_;
    unsigned long long int *edge_access_time_;
    int32_t train_step_;
    int32_t iter_;

    int32_t max_ids_; // for allocating feature buffer

    bght::bcht<int32_t, int32_t> *node_map_;
    bght::bcht<int32_t, int32_t> *pos_map_;

    bght::bcht<int32_t, char> *edge_index_map_;
    bght::bcht<int32_t, int32_t> *edge_offset_map_;

    int32_t node_capacity_;
    int32_t edge_capacity_;

    int32_t *cache_ids_;
    int32_t *cache_offset_;
    pair_type *pair_;
    pair_type *graph_pair_;

    int32_t *d_global_count_;
    int32_t *h_global_count_;
    int32_t h_cache_hit_;
    int32_t find_iter_;

    int32_t *start_ptr_;
    int32_t *stop_ptr_;
};

CacheController *NewPreSCCacheController(int32_t train_step, int32_t device_count)
{
    return new PreSCCacheController(train_step, device_count);
}

void UnifiedCache::Initialize(
    int64_t cache_memory,
    int32_t float_feature_len,
    int32_t train_step,
    int32_t device_count,
    int32_t cpu_cache_capacity,
    int32_t gpu_cache_capacity,
    std::string dataset_path)
{
    device_count_ = device_count;
    cache_controller_.resize(device_count_);
    for (int32_t i = 0; i < device_count_; i++)
    {
        CacheController *cctl = NewPreSCCacheController(train_step, device_count_);
        cache_controller_[i] = cctl;
    }
    // std::cout << "Cache Controler Initialize\n";

    if (float_feature_len > 0)
    {
        float_feature_cache_.resize(device_count_);
    }
    cudaCheckError();

    cache_memory_ = cache_memory;
    float_feature_len_ = float_feature_len;
    cpu_cache_capacity_ = cpu_cache_capacity;
    gpu_cache_capacity_ = gpu_cache_capacity;
    is_presc_ = true;
    dataset_path_ = dataset_path;
}

void UnifiedCache::InitializeCacheController(
    int32_t dev_id,
    int32_t total_num_nodes)
{
    cache_controller_[dev_id]->Initialize(dev_id, total_num_nodes);
}

void UnifiedCache::Finalize()
{
    if (!destroy_file((dataset_path_ + "accesstimes").c_str())) {
        std::cerr << "Access Time Delete failed\n";
    }
}

void UnifiedCache::FindFeat(
    int32_t *sampled_ids,
    int32_t *cache_offset,
    int32_t *node_counter,
    int32_t op_id,
    void *stream,
    int32_t dev_id)
{
    cache_controller_[dev_id]->FindFeat(sampled_ids, cache_offset, node_counter, op_id, stream);
}

void UnifiedCache::FindTopo(
    int32_t *input_ids,
    char *partition_index,
    int32_t *partition_offset,
    int32_t batch_size,
    int32_t op_id,
    void *strm_hdl,
    int32_t dev_id)
{
    cache_controller_[dev_id]->FindTopo(input_ids, partition_index, partition_offset, batch_size, op_id, strm_hdl, dev_id);
}

void UnifiedCache::HybridInit(FeatureStorage *feature, GraphStorage *graph)
{ // single gpu

    hipHostAlloc(&cpu_float_features_, int64_t(int64_t(cpu_cache_capacity_) * float_feature_len_ * sizeof(float)), hipHostMallocMapped);

    // std::cout << "Start selecting cache candidates\n";
    std::vector<unsigned long long int *> node_access_time;
    for (int32_t i = 0; i < device_count_; i++)
    {
        hipSetDevice(i);
        node_access_time.push_back(cache_controller_[i]->GetNodeAccessedMap());
    }
    // 拿到全局热度global_node_access_time
    int32_t total_num_nodes = feature->TotalNodeNum();
    unsigned long long int *global_node_access_time;
    hipMalloc(&global_node_access_time, total_num_nodes * sizeof(unsigned long long int));
    hipMemset(global_node_access_time, 0, total_num_nodes * sizeof(unsigned long long int));

    for (int32_t i = 0; i < device_count_; i++)
    {
        hipSetDevice(i);
        // 假设每个 GPU 的 node_access_time 已经是设备指针
        accumulate<<<(total_num_nodes + 1023) / 1024, 1024>>>(global_node_access_time, node_access_time[i], total_num_nodes);
        hipDeviceSynchronize();
        cudaCheckError();
    }
    unsigned long long* h_node_access_time = (unsigned long long*)malloc(int64_t(int64_t(total_num_nodes) * sizeof(unsigned long long int)));
    hipMemcpy(h_node_access_time, global_node_access_time, int64_t(int64_t(total_num_nodes) * sizeof(unsigned long long int)), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    if (!dump_access_times_to_txt((dataset_path_ + "accesstimes").c_str(), h_node_access_time,
                                total_num_nodes)) {
        std::cerr << "Dump failed!\n";
        std::free(h_node_access_time);
        return;
    }

    // std::cout << "Successfully wrote " << total_num_nodes
    //         << " lines to " << dataset_path_.c_str() << '\n';

    std::free(h_node_access_time);

    // for (int i = 0; i < device_count_; i++)
    // {
    //     hipSetDevice(i);
    //     cudaCheckError();
    //     int32_t total_num_nodes = feature->TotalNodeNum();
    //     total_num_nodes_ = total_num_nodes;
    //     int32_t *node_cache_order;
    //     hipMalloc(&node_cache_order, int64_t(int64_t(total_num_nodes) * sizeof(int32_t)));
    //     cudaCheckError();
    //     init_cache_order<<<80, 1024>>>(node_cache_order, total_num_nodes);
    //     thrust::sort_by_key(thrust::device, global_node_access_time, global_node_access_time + total_num_nodes, node_cache_order, thrust::greater<unsigned long long int>());
    //     cudaCheckError();
    //     QF_.push_back(node_cache_order);
    // }

    for(int32_t i = 0; i < device_count_; i++){
        hipSetDevice(i);
        cudaCheckError();
        int32_t total_num_nodes = feature->TotalNodeNum();
        total_num_nodes_ = total_num_nodes;
        int32_t* node_cache_order;
        // hipMalloc(&node_cache_order, int64_t(int64_t(total_num_nodes) * sizeof(int32_t)));
        hipHostAlloc(&node_cache_order, int64_t(int64_t(total_num_nodes) * sizeof(int32_t)),hipHostMallocMapped);
        cudaCheckError();
        init_cache_order<<<80, 1024>>>(node_cache_order, total_num_nodes);
        thrust::sort_by_key(thrust::device, node_access_time[i], node_access_time[i] + total_num_nodes, node_cache_order, thrust::greater<unsigned long long int>());
        cudaCheckError();
        QF_.push_back(node_cache_order);
    }

    for (int i = 0; i < device_count_; i++)
    {
        cache_controller_[i]->InitializeMap(gpu_cache_capacity_ + cpu_cache_capacity_, 100);
        cache_controller_[i]->HybridInsert(QF_[i], cpu_cache_capacity_, gpu_cache_capacity_);
    }

    d_float_feature_cache_ptr_.resize(device_count_);

    for (int32_t i = 0; i < device_count_; i++)
    {
        hipSetDevice(i);
        // float** new_ptr;
        // hipMalloc(&new_ptr, 1 * sizeof(float*));
        // d_float_feature_cache_ptr_[i] = new_ptr;

        if (float_feature_len_ > 0)
        {
            float *new_float_feature_cache;
            hipMalloc(&new_float_feature_cache, int64_t(int64_t(int64_t(gpu_cache_capacity_) * float_feature_len_) * sizeof(float)));
            // std::cout << "Allocate GPU Feature Cache" << gpu_cache_capacity_ << "\n";
            // FeatFillUp<<<128, 1024>>>(gpu_cache_capacity_, float_feature_len_, new_float_feature_cache, cpu_float_feature, QF_[i], Kg_, j);
            float_feature_cache_[i] = new_float_feature_cache;
            // init_feature_cache<<<1,1>>>(d_float_feature_cache_ptr_[i], new_float_feature_cache, i);          //j: device id in clique
            // cudaCheckError();
        }
    }

    hipDeviceSynchronize();
    is_presc_ = false;

    // std::cout << "Finish load feature cache\n";
}

int32_t UnifiedCache::NodeCapacity(int32_t dev_id)
{
    return node_capacity_[dev_id / Kg_];
}

int32_t UnifiedCache::CPUCapacity()
{
    return cpu_cache_capacity_;
}

int32_t UnifiedCache::GPUCapacity()
{
    return gpu_cache_capacity_; // single gpu version
}

float *UnifiedCache::Float_Feature_Cache(int32_t dev_id)
{
    return float_feature_cache_[dev_id];
}

float **UnifiedCache::Global_Float_Feature_Cache(int32_t dev_id)
{
    return d_float_feature_cache_ptr_[dev_id];
}

int32_t UnifiedCache::MaxIdNum(int32_t dev_id)
{
    return cache_controller_[dev_id]->MaxIdNum();
}

unsigned long long int *UnifiedCache::GetEdgeAccessedMap(int32_t dev_id)
{
    return cache_controller_[dev_id]->GetEdgeAccessedMap();
}

void UnifiedCache::CacheProfiling(
    int32_t *sampled_ids,
    int32_t *agg_src_id,
    int32_t *agg_dst_id,
    int32_t *agg_src_off,
    int32_t *agg_dst_off,
    int32_t *node_counter,
    int32_t *edge_counter,
    void *stream,
    int32_t dev_id)
{
    cache_controller_[dev_id]->CacheProfiling(sampled_ids, agg_src_id, agg_dst_id, agg_src_off, agg_dst_off, node_counter, edge_counter, is_presc_, stream);
}

void UnifiedCache::AccessCount(
    int32_t *d_key,
    int32_t num_keys,
    void *stream,
    int32_t dev_id)
{
    cache_controller_[dev_id]->AccessCount(d_key, num_keys, stream);
}

void UnifiedCache::FeatCacheLookup(int32_t *sampled_ids, int32_t *cache_index,
                                   int32_t *node_counter, float *dst_float_buffer,
                                   int32_t op_id, int32_t dev_id, hipStream_t strm_hdl)
{
    dim3 block_num(64, 1);
    dim3 thread_num(1024, 1);
    // float** gpu_float_feature     = Global_Float_Feature_Cache(dev_id);
    int32_t cpu_cache_capacity = CPUCapacity();
    int32_t gpu_cache_capacity = GPUCapacity();
    // printf("test_lookup\n");
    feat_cache_lookup<<<block_num, thread_num, 0, (strm_hdl)>>>(
        cpu_float_features_, float_feature_cache_[dev_id], float_feature_len_,
        sampled_ids, cache_index,
        cpu_cache_capacity, gpu_cache_capacity,
        node_counter, dst_float_buffer,
        op_id);
}
